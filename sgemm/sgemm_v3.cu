// optimize sgemm

#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

// K: ldA
// N: ldB
template <
    const int BLOCK_SIZE_M /* height of block of C that each thread block calculate //128 */,  
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory //8
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate //128
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate //8
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate //8
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void Sgemm( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;//BlkDim.x 16
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;//BlkDim.y 16
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;//BTBT 一个blk有256=16*16个thrd

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];//[2][8][128]
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X];//[128][128]
    #pragma unroll
    for(int i=0; i<THREAD_SIZE_Y; i++){
        #pragma unroll
        for(int j=0; j<THREAD_SIZE_X; j++){
            accum[i][j]=0.0;
        }
    }
    
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);//BTBT ??? 4是指float4?
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];//BTBT (128*8)/256=4 每个thrd负责取多少glb的elem
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row (in BlkTil) //BTBT ??? 除4啥意思???每个thrd取一次,每次一个float4?
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;//BTBT 每thrd每次取float4的话,每个blk的一行elem需要多少thrd就能一次取完 8/4=2
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;//BTBT 该thrd在BlkTIl内是第几行
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; //BTBT 该thrd在BlkTIl内是第几列
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;//BTBT 一个BlkTil内有多少thrd row 256/2=128
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];//BTBT 指针移动到该BlkTil在A中的左上角起点
    B = &B[BLOCK_SIZE_N * bx];//BTBT 指针移动到该BlkTil在B中的左上角起点

    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {//BTBT 当前场景下共256thrd,每行2thrd,有128行,256thrd每thrd取一个float4填入SMEM,刚好填满As[0]which is [4*2][128]
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(  //OFFSET(row, col, ld) ((row) * (ld) + (col))
            A_TILE_ROW_START + i, // row (tid/A_TILE_THREAD_PER_ROW=tid/2)+i
            A_TILE_COL, // col tid % A_TILE_THREAD_PER_ROW * 4
            K )]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index]; //BTBT ??? 写SMEM是否有bank conflic
        As[0][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();

    // registers for A and B SMEM
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];

    //load index of the tile
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    const int tile_index = (warp_id%4)*16 + (lane_id/16)*8 + (lane_id%2)*4;

    // load A from shared memory to register
    FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[0][0][tile_index]);//BTBT 参考https://zhuanlan.zhihu.com/p/481600052 ,这里是取128的前半,下面取后半
    FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[0][0][tile_index + 64]);
    // load B from shared memory to register
    FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[0][0][tile_index]);
    FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[0][0][tile_index + 64]);

    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;
        int next_stage_flag = load_stage_idx;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K; ++j){
            // load next tile from shared mem to register //BTBT ??? 这个load和write的次序貌似错了?tile_index的计算对于A,B来说应该不一样,否则无法计算完整矩阵?这个文件问题较多,不建议看
            next_stage_flag = (j==BLOCK_SIZE_K-1)?load_stage_idx:write_stage_idx;
            // load A from shared memory to register
            FETCH_FLOAT4(frag_a[(j+1)%2][0]) = FETCH_FLOAT4(As[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index]);
            FETCH_FLOAT4(frag_a[(j+1)%2][4]) = FETCH_FLOAT4(As[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index + 64]);
            // load B from shared memory to register
            FETCH_FLOAT4(frag_b[(j+1)%2][0]) = FETCH_FLOAT4(Bs[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index]);
            FETCH_FLOAT4(frag_b[(j+1)%2][4]) = FETCH_FLOAT4(Bs[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index + 64]);
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }
    }while(tile_idx< K);

    //store C00 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i,
        BLOCK_SIZE_N * bx + tx * 4,
        N)]) = FETCH_FLOAT4(accum[i][0]);
    }
    //store C01 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i,
        BLOCK_SIZE_N * bx + tx * 4 + 64,
        N)]) = FETCH_FLOAT4(accum[i][4]);
    }
    //store C10 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i + 64,
        BLOCK_SIZE_N * bx + tx * 4,
        N)]) = FETCH_FLOAT4(accum[i+4][0]);
    }
    //store C11 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i+ 64,
        BLOCK_SIZE_N * bx + tx * 4 + 64,
        N)]) = FETCH_FLOAT4(accum[i+4][4]);
    }
}

int main(int argc, char** argv) {
    if (argc != 4) {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_B = sizeof(float) * K * N;
    size_t bytes_C = sizeof(float) * M * N;
    float* h_A = (float*)malloc(bytes_A);
    float* h_B = (float*)malloc(bytes_B);
    float* h_C = (float*)malloc(bytes_C);
    float* h_C1 = (float*)malloc(bytes_C);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_B, bytes_B));
    checkCudaErrors(hipMalloc(&d_C, bytes_C));
    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    // don't edit it
    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;
    const bool ENABLE_DOUBLE_BUFFER = false;
    int k_block = K / BLOCK_SIZE_K;
    int stride = 2;

    // 生成A的数据
    for( int i = 0; i < M * K; i++ ) {
        int row = (i / K);
        int col = (i % K);
        int row_block = row / BLOCK_SIZE_M;
        int col_block = col / BLOCK_SIZE_K;
        if ((row_block * k_block + col_block) % stride == 0) h_A[i] = 1;
        else {
            h_A[i] = 0;
        }
    }

    // 生成B的数据
    for( int i = 0; i < K * N; i++ ) {
        if ( i >= K * N / 2) h_B[i] = 2;
        else {
            h_B[i] = 0;
        }
    }

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes_B, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 1000;

    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        Sgemm<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy( h_C, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // cublas
    
    hipblasHandle_t blas_handle;  
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        hipblasSgemm (blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
            M, N, K, &alpha, 
            d_A, K, d_B, N, &beta, d_C, N
        );
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf( "CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);

    hipblasDestroy(blas_handle); 

    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C[i] - h_C1[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C[i], h_C1[col * M + row], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}
