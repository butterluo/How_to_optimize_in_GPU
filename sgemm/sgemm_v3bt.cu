// optimize sgemm

#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

// K: ldA
// N: ldB
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
    > 
__global__ void Sgemm( 
    float * __restrict__ A,
    float * __restrict__ B,
    float * __restrict__ C, 
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X];
    #pragma unroll
    for(int i=0; i<THREAD_SIZE_Y; i++){//BTBT ??? 考虑用汇编置零,或不置零
        #pragma unroll
        for(int j=0; j<THREAD_SIZE_X; j++){
            accum[i][j]=0.0;
        }
    }
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory//BTBT ??? 要从glb到reg再到smem,这里是暂存用的reg,是否考虑汇编或txture
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);//BTBT A:每个thrd加载多少个四元组
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4*ldg_num_a];//A临时存放每个thrd从glb加载过来的数据
    float ldg_b_reg[4*ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;//A在行维度需要有多少thrd,每个thrd每次加载float4
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;//A在行维度需要A_TILE_THREAD_PER_ROW个thrd的情况下,这个thrd属于第几行
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4; //A在行维度需要A_TILE_THREAD_PER_ROW个thrd的情况下,这个thrd属于第几列
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;//A在行维度需要A_TILE_THREAD_PER_ROW个thrd的情况下,一个thrdBlk可安排多少行
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by)* K];//BTBT bug 没判断有没超范围或padding
    B = &B[BLOCK_SIZE_N * bx];

    //load index of the tile
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    const int tile_index = (warp_id%4)*16 + (lane_id/16)*8 + (lane_id%2)*4;//BTBT bug  B的tile_index的计算可能不是这样的

    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K )]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];    //BTBT bug 这里貌似也有bank conflict,同wrp的lane0和lane1写同一个bank ???
        As[0][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
    }
    // load B from global memory to shared memory
    #pragma unroll
    for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
                B_TILE_ROW_START + i, // row
                B_TILE_COL, // col
                N )]);
    }
    __syncthreads();

    // load A from shared memory to register
    FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[0][0][tile_index]);      //当wrpId和laneId不断变化时,其实是在同一行取数,warp0和warp4取得是同样的16个元素。而warp里面，线程0、2、4、6、8、10、12、14是取得同样的4个元素。由于取得是同样的元素，同一个bank触发多播的机制，
    FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[0][0][tile_index + 64]);
    // load B from shared memory to register
    FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[0][0][tile_index]);
    FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[0][0][tile_index + 64]);

    int write_stage_idx = 1;
    int tile_idx = 0;
    do{
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if(tile_idx< K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K )]);
            }
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N )]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;
        int next_stage_flag = load_stage_idx;

        #pragma unroll
        for(int j=0; j<BLOCK_SIZE_K; ++j){
            // load next tile from shared mem to register 
            next_stage_flag = (j==BLOCK_SIZE_K-1)?load_stage_idx:write_stage_idx;//BTBT bug 这里的next_stage_flag要一直等于load_stage_idx吧,write_stage_idx是下一轮才写进smem的吧 ???
            // load A from shared memory to register //BTBT bug 第一轮时貌似不能这样做,因为As[1]和Bs[1]中没有任何数据
            FETCH_FLOAT4(frag_a[(j+1)%2][0]) = FETCH_FLOAT4(As[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index]);
            FETCH_FLOAT4(frag_a[(j+1)%2][4]) = FETCH_FLOAT4(As[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index + 64]);
            // load B from shared memory to register
            FETCH_FLOAT4(frag_b[(j+1)%2][0]) = FETCH_FLOAT4(Bs[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index]);
            FETCH_FLOAT4(frag_b[(j+1)%2][4]) = FETCH_FLOAT4(Bs[next_stage_flag][(j+1)%BLOCK_SIZE_K][tile_index + 64]);
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
            #pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
                #pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j%2][thread_y] * frag_b[j%2][thread_x];
                }
            }
        }

        if(tile_idx < K){
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_M ; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW_START + i]=ldg_a_reg[ldg_index+3];
            }
            // load B from global memory to shared memory
            #pragma unroll
            for ( int i = 0 ; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }
    }while(tile_idx< K);

    //store C00 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i,
        BLOCK_SIZE_N * bx + tx * 4,
        N)]) = FETCH_FLOAT4(accum[i][0]);
    }
    //store C01 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i,
        BLOCK_SIZE_N * bx + tx * 4 + 64,
        N)]) = FETCH_FLOAT4(accum[i][4]);
    }
    //store C10 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i + 64,
        BLOCK_SIZE_N * bx + tx * 4,
        N)]) = FETCH_FLOAT4(accum[i+4][0]);
    }
    //store C11 block
    for(int i=0; i<4; i++){
      FETCH_FLOAT4(C[OFFSET(
        BLOCK_SIZE_M * by + ty * 4 + i+ 64,
        BLOCK_SIZE_N * bx + tx * 4 + 64,
        N)]) = FETCH_FLOAT4(accum[i+4][4]);
    }
}

int main(int argc, char** argv) {
    if (argc != 4) {
        printf("usage: ./main [M] [K] [N]\n");
        exit(0);
    }
    size_t M = atoi(argv[1]);
    size_t K = atoi(argv[2]);
    size_t N = atoi(argv[3]);

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_B = sizeof(float) * K * N;
    size_t bytes_C = sizeof(float) * M * N;
    float* h_A = (float*)malloc(bytes_A);
    float* h_B = (float*)malloc(bytes_B);
    float* h_C = (float*)malloc(bytes_C);
    float* h_C1 = (float*)malloc(bytes_C);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_B, bytes_B));
    checkCudaErrors(hipMalloc(&d_C, bytes_C));
    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    // don't edit it //BTBT ???
    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;
    const bool ENABLE_DOUBLE_BUFFER = false;
    int k_block = K / BLOCK_SIZE_K;
    int stride = 2;

    // 生成A的数据
    for( int i = 0; i < M * K; i++ ) {
        int row = (i / K);
        int col = (i % K);
        int row_block = row / BLOCK_SIZE_M;
        int col_block = col / BLOCK_SIZE_K;
        if ((row_block * k_block + col_block) % stride == 0) h_A[i] = 1;
        else {
            h_A[i] = 0;
        }
    }

    // 生成B的数据
    for( int i = 0; i < K * N; i++ ) {
        if ( i >= K * N / 2) h_B[i] = 2;
        else {
            h_B[i] = 0;
        }
    }

    checkCudaErrors(hipMemcpy( d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy( d_B, h_B, bytes_B, hipMemcpyHostToDevice));
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 1000;

    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        Sgemm<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER> 
        <<< dimGrid, dimBlock >>>(d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy( h_C, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // cublas
    
    hipblasHandle_t blas_handle;  
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy( d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0 ; run < nIter; run ++ ) {
        hipblasSgemm (blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
            M, N, K, &alpha, 
            d_A, K, d_B, N, &beta, d_C, N
        );
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy( h_C1, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf( "CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);

    hipblasDestroy(blas_handle); 

    
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C[i] - h_C1[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_C[i], h_C1[col * M + row], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);
    
    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}
/**
 * TODO:
 * 没整除时如何padding
 * 对half8的支持
 * 实习生测试那个sample快
 * */